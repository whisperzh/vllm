#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

extern "C" {
    // Function to open IPC handle and return a pointer to the device memory
    void* open_ipc_handle(const void* handle_bytes) {
        hipIpcMemHandle_t handle;
        // Copy handle_bytes to handle
        memcpy(&handle, handle_bytes, sizeof(hipIpcMemHandle_t));

        void* device_ptr;
        hipError_t err = hipIpcOpenMemHandle(&device_ptr, handle, hipIpcMemLazyEnablePeerAccess);
        if (err != hipSuccess) {
            std::cerr << "Failed to open IPC handle: " << hipGetErrorString(err) << std::endl;
            return nullptr;
        }

        return device_ptr;
    }
}