#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <filesystem>

extern "C" {
    // Function to open IPC handle and return a pointer to the device memory
    void* open_ipc_handle(const char* handle_file) {
        hipIpcMemHandle_t handle;
        std::ifstream file(handle_file, std::ios::binary);
        if (!file.is_open()) {
            std::cerr << "Failed to open handle file: " << handle_file << std::endl;
            return nullptr;
        }
        file.read(reinterpret_cast<char*>(&handle), sizeof(hipIpcMemHandle_t));
        file.close();

        float* device_ptr;
        hipError_t err = hipIpcOpenMemHandle((void**)&device_ptr, handle, hipIpcMemLazyEnablePeerAccess);
        if (err != hipSuccess) {
            std::cerr << "Failed to open IPC handle: " << hipGetErrorString(err) << std::endl;
            return nullptr;
        }

        return device_ptr;
    }
}